#include "hip/hip_runtime.h"
#include <cstdio>
#include "CR_Device_functions.cuh"

__global__ void list_print(int nmax, float * in) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    printf("Thread %i shows %f \n", i, in[i]);
}


__global__ void CR_Kernel_Forward(
    float * alist, float * blist, float * clist, float * dlist,
    int stride, int DMax) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int idx = stride * i;

    int pre_idx = idx - stride/2;
    int nex_idx = idx + stride/2;

    float a[] = {0.0f, 0.0f, 0.0f};
    float b[] = {0.0f, 0.0f, 0.0f};
    float c[] = {0.0f, 0.0f, 0.0f};
    float d[] = {0.0f, 0.0f, 0.0f};

    float k1, k2;

    a[1] = alist[idx];
    b[1] = blist[idx];
    c[1] = clist[idx];
    d[1] = dlist[idx];

    if (idx == 0) {

        k1 = 0.0f;
        a[2] = alist[nex_idx];
        b[2] = blist[nex_idx];
        c[2] = clist[nex_idx];
        d[2] = dlist[nex_idx];
        k2 = c[1]/b[2];


    } else if (0 == (DMax-1-i*2) ) {

        k2 = 0.0f;
        a[0] = alist[pre_idx];
        b[0] = blist[pre_idx];
        c[0] = clist[pre_idx];
        d[0] = dlist[pre_idx];
        k1 = a[1]/b[0];

    } else {

        a[0] = alist[pre_idx];
        b[0] = blist[pre_idx];
        c[0] = clist[pre_idx];
        d[0] = dlist[pre_idx];
        a[2] = alist[nex_idx];
        b[2] = blist[nex_idx];
        c[2] = clist[nex_idx];
        d[2] = dlist[nex_idx];

        k1 = a[1]/b[0];
        k2 = c[1]/b[2];

    }

    alist[idx] = -a[0]*k1;
    blist[idx] = b[1] - c[0]*k1 - a[2]*k2;
    clist[idx] = -c[2]*k2;
    dlist[idx] = d[1] - d[0]*k1 - d[2]*k2;

}

__global__ void Solve2By2(
    float * alist, float * blist, float * clist, float * dlist, float * xlist,
    int stride ) {

    int i = blockIdx.x*blockDim.x + threadIdx.x; //i = 0 or 1

    if (i == 0) {
        float k = clist[0]/blist[stride];
        xlist[0] = (dlist[0]-dlist[stride]*k)/(blist[0]-alist[stride]*k);
    } else {
        float k = blist[0]/alist[stride];
        xlist[stride] = (dlist[0]-dlist[stride]*k)/(clist[0]-blist[stride]*k);
    }

}

__global__ void CR_Kernel_Backward(
    float * alist, float * blist, float * clist, float * dlist, float * xlist,
    int stride, int DMax) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int idx = stride * (2*i+1);

    float xupper = xlist[idx - stride];
    float xlower = 0.0f;
    if (idx != DMax) {
        //printf("i = %i, idx = %i\n", i, idx);
        xlower = xlist[idx + stride];
    }

    xlist[idx] = (dlist[idx] - alist[idx]*xupper - clist[idx]*xlower)/blist[idx];

}
