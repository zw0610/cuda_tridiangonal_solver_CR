#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cstdlib>

#include "CR_Class.h"

int main( ) {

    size_t diagonal_size = 24;

    CR_Solver crs = CR_Solver(diagonal_size);

    //Generate sampel data
    srand (time(NULL));

    thrust::device_vector<float> alist(diagonal_size);
    thrust::device_vector<float> blist(diagonal_size);
    thrust::device_vector<float> clist(diagonal_size);
    thrust::device_vector<float> dlist(diagonal_size);
    thrust::device_vector<float> xlist(diagonal_size);

    float * ptr_alist = thrust::raw_pointer_cast(alist.data());
    float * ptr_blist = thrust::raw_pointer_cast(blist.data());
    float * ptr_clist = thrust::raw_pointer_cast(clist.data());
    float * ptr_dlist = thrust::raw_pointer_cast(dlist.data());
    float * ptr_xlist = thrust::raw_pointer_cast(xlist.data());

    for (int i=0; i < diagonal_size; i++) {
        alist[i] = i+2;
        blist[i] = i+1;
        clist[i] = i+3;
        dlist[i] = i+10;//rand() % 100 + 1;
        xlist[i] = 0.0f;
    }

    alist[0] = float(0.0);
    clist[diagonal_size-1] = float(0.0);

    crs.Solve(ptr_alist, ptr_blist, ptr_clist, ptr_dlist, ptr_xlist);

    for (auto item : xlist) {
        std::cout << item << std::endl;
    }

    return 0;

}
