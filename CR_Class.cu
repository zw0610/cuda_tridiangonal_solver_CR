#include "CR_Class.h"
#include "CR_Device_functions.cuh"

CR_Solver::CR_Solver(int coming_ds) {

    diagonal_size = coming_ds;

    int stride = 0;
    int current_size = diagonal_size;
    for (stride=2; stride <diagonal_size; stride *= 2) {
        sdlist.push_back(current_size);
        current_size = (current_size+1)/2;
    }
    sdlist.push_back(2);

}


void CR_Solver::Solve(float * alist, float * blist, float * clist, float * dlist, float * xlist) {

    int next_size;
    int current_size = diagonal_size;
    int stride=2;
    for (; stride <diagonal_size; stride *= 2) {
        next_size = (current_size+1)/2;
        CR_Kernel_Forward<<<1,next_size>>>(alist, blist, clist, dlist, stride, current_size);
        //hipDeviceSynchronize();
        current_size = next_size;
    }

    Solve2By2<<<1,2>>>(alist, blist, clist, dlist, xlist, stride/2);

    int neo_stride = stride/4;
    for (int id = (sdlist.size()-1); id>0; id--) {
        int even_number = sdlist[id-1] - sdlist[id];
        int last_index = (sdlist[id-1]-1)*neo_stride;
        CR_Kernel_Backward<<<1,even_number>>>(alist, blist, clist, dlist, xlist, neo_stride, last_index);
        neo_stride = neo_stride/2;
    }



}
